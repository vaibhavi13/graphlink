#include <hip/hip_runtime.h>
#include <stdio.h>

#define BLOCK_SIZE 256

extern "C" __global__ void kmeans_kernel(float* points, float* centroids, int* cluster_assignments, int k, int d, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        float min_dist = 10000000000;
        int min_idx = -1;
        for (int i = 0; i < k; i++) {
            float dist = 0.0f;
            for (int j = 0; j < d; j++) {
                float diff = points[idx * d + j] - centroids[i * d + j];
                dist += diff * diff;
            }
            if (dist < min_dist) {
                min_dist = dist;
                min_idx = i;
            }
        }
        cluster_assignments[idx] = min_idx;
    }
}


